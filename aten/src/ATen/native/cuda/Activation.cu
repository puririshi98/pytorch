#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include <ATen/native/Activation.h>

#include <cmath>

#include <thrust/tuple.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/core/Array.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/DistributionTemplates.h>
#include <c10/cuda/CUDAMathCompat.h>

namespace at {
namespace native {

// -----------------------------------
// prelu forward
// -----------------------------------
template <typename scalar_t>
void prelu_cuda_kernel_share_weights(
  const Tensor& input,
  Tensor& result,
  const scalar_t* weight_data)
{
  auto iter = TensorIterator::unary_op(result, input);

  at::native::gpu_kernel(iter,
    [weight_data] GPU_LAMBDA (scalar_t input_val) {
        return (input_val > 0) ? input_val : *weight_data * input_val;
    });
}

template <typename scalar_t>
__global__ void prelu_cuda_kernel_multi_weights(
  scalar_t* result_data,
  const scalar_t* input_data,
  const scalar_t* weight_data,
  int64_t input_stride0,
  int64_t input_stride1,
  int64_t input_numel) {

  int64_t linearId = blockIdx.x * blockDim.x + threadIdx.x;
  if (linearId >= input_numel) return;

  // multiply values at each channel with weight[channel_index]
  int64_t channel = (linearId % input_stride0) / input_stride1;
  scalar_t input_data_val = input_data[linearId];
  result_data[linearId] = (input_data_val > 0) ? input_data_val : weight_data[channel] * input_data_val;
}

Tensor prelu_cuda(const Tensor& self, const Tensor& weight_) {
  TORCH_CHECK(self.is_cuda());
  TORCH_CHECK(weight_.is_cuda());

  auto input = self.contiguous();
  auto weight = weight_.contiguous();

  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(weight.is_contiguous());

  int64_t weight_num = weight.numel();
  Tensor result = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto strides = input.strides();

  // case1: shared weight for all channels
  if (weight_num == 1) {
    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_cuda", [&] {
      prelu_cuda_kernel_share_weights<scalar_t>(
        input,
        result,
        weight.data_ptr<scalar_t>());
    });
  }
  else { // case2: multiple weights, one for each channel
    int64_t input_ndim = input.dim();
    TORCH_CHECK(input_ndim > 0, "Not allow zero-dim input tensor.");

    int64_t channel_size = 1; // channel_size default to 1
    int64_t input_stride0 = 1, input_stride1 = 1;

    if (input_ndim > 1) {
      channel_size = input.size(1); // channel is the 2nd dim of input
      input_stride0 = strides[0];
      input_stride1 = strides[1];
    }
    TORCH_CHECK(channel_size == weight_num,
      "Mismatch of parameter numbers and input channel size. Found parameter numbers = ", weight_num,
      " and channel size = ", channel_size, ".");

    // config to run cuda kernel
    int64_t input_numel = input.numel();
    const dim3 block = dim3(std::min(static_cast<int64_t>(cuda::getApplyBlock().x), input_numel));
    dim3 grid;
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
    TORCH_CHECK(cuda::getApplyGrid(input_numel, grid, curDevice), "prelu: input too large or too many dimensions");

    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_cuda", [&] {
      prelu_cuda_kernel_multi_weights<scalar_t>
      <<<grid, block, 0, stream>>>(
        result.data_ptr<scalar_t>(),
        input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        input_stride0,
        input_stride1,
        input_numel);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
  }
  return result;
}

// -----------------------------------
// prelu backward
// -----------------------------------
template <typename scalar_t>
void prelu_cuda_backward_kernel_share_weights(
  const Tensor& input,
  const Tensor& grad_out,
  Tensor& input_grad,
  Tensor& weight_grad_collector,
  const scalar_t* weight_data) {
  at::TensorIterator iter = TensorIteratorConfig()
      .add_output(input_grad)
      .add_output(weight_grad_collector)
      .add_input(input)
      .add_input(grad_out)
      .build();

  // N.B. `std::tuple` does not support `::operator=` on device code.
  gpu_kernel_multiple_outputs(iter, [=] GPU_LAMBDA (scalar_t input, scalar_t grad_out) -> thrust::tuple<scalar_t, scalar_t> {
    scalar_t input_grad = input > 0 ? grad_out : (*weight_data) * grad_out;
    scalar_t weight_grad_collector = input > 0 ? scalar_t(0) : input * grad_out;
    return {input_grad, weight_grad_collector};
  });
}

template <typename scalar_t>
__global__ void prelu_cuda_backward_kernel_multi_weights(
  const scalar_t* input_data,
  const scalar_t* weight_data,
  const scalar_t* grad_out_data,
  scalar_t* input_grad_data,
  scalar_t* weight_grad_collector,
  int64_t input_stride0,
  int64_t input_stride1,
  int64_t input_numel) {

  int64_t linearId = blockIdx.x * blockDim.x + threadIdx.x;
  if (linearId >= input_numel) return;
  int64_t channel = (linearId % input_stride0) / input_stride1;
  scalar_t input_data_val = input_data[linearId];
  scalar_t grad_out_data_val = grad_out_data[linearId];
  input_grad_data[linearId] = (input_data_val > 0) ? grad_out_data_val : weight_data[channel] * grad_out_data_val;
  weight_grad_collector[linearId] = (input_data_val > 0) ? scalar_t(0) : input_data_val * grad_out_data_val;
}

std::tuple<Tensor, Tensor> prelu_backward_cuda(const Tensor& grad_out_, const Tensor& self, const Tensor& weight_) {
  TORCH_CHECK(grad_out_.is_cuda());
  TORCH_CHECK(self.is_cuda());
  TORCH_CHECK(weight_.is_cuda());

  auto input = self.contiguous();
  auto grad_out = grad_out_.contiguous();
  auto weight = weight_.contiguous();

  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(weight.is_contiguous());
  TORCH_CHECK(grad_out.is_contiguous());

  int64_t weight_num = weight.numel();
  auto strides = input.strides();
  auto dims = input.dim();
  Tensor input_grad = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor weight_grad = at::empty_like(weight, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor weight_grad_collector = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  // case1: shared parameter for all channels
  if (weight_num == 1) {
    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_backward_cuda", [&] {
      prelu_cuda_backward_kernel_share_weights<scalar_t>(
        input,
        grad_out,
        input_grad,
        weight_grad_collector,
        weight.data_ptr<scalar_t>());
    });
    weight_grad.fill_(weight_grad_collector.sum());
  }
  else { // case2: multiple parameters, one for each channel
    int64_t input_ndim = input.dim();
    TORCH_CHECK(input_ndim > 0, "Not allow zero-dim input tensor.");

    int64_t channel_size = 1; // channel_size default to 1
    int64_t input_stride0 = 1, input_stride1 = 1;

    if (input_ndim > 1) {
      channel_size = input.size(1); // channel is the 2nd dim of input
      input_stride0 = strides[0];
      input_stride1 = strides[1];
    }
    TORCH_CHECK(channel_size == weight_num,
      "Mismatch of parameter numbers and input channel size. Found parameter numbers = ", weight_num,
      " and channel size = ", channel_size, ".");

    // config to run cuda kernel
    int64_t input_numel = input.numel();
    const dim3 block = dim3(std::min(static_cast<int64_t>(cuda::getApplyBlock().x), input_numel));
    dim3 grid;
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
    TORCH_CHECK(cuda::getApplyGrid(input_numel, grid, curDevice), "prelu_backward_cuda: input too large or too many dimensions");

    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_backward_cuda", [&] {
      prelu_cuda_backward_kernel_multi_weights<scalar_t>
      <<<grid, block, 0, stream>>>(
        input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        grad_out.data_ptr<scalar_t>(),
        input_grad.data_ptr<scalar_t>(),
        weight_grad_collector.data_ptr<scalar_t>(),
        input_stride0,
        input_stride1,
        input_numel);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
    // update weight_grad
    std::vector<int64_t> reduce_dims;
    reduce_dims.push_back(0);
    if (dims > 2) {
      for(int64_t i = 2; i < dims; i++) reduce_dims.push_back(i);
    }
    weight_grad = weight_grad_collector.sum(reduce_dims);
  }
  return std::tuple<Tensor, Tensor>{input_grad, weight_grad};
}

// -----------------------------------
// rrelu
// -----------------------------------
template <typename scalar_t, int unroll_factor, typename F>
#if __CUDA_ARCH__ >= 350 || defined __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_2(256, 4)
#endif
__global__ void rrelu_with_noise_cuda_kernel(
    int numel,
    PhiloxCudaState philox_args,
    scalar_t* output,
    scalar_t* input,
    scalar_t* noise,
    double lower,
    double upper,
    const F& random_func) {
  auto seeds = at::cuda::philox::unpack(philox_args);
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(std::get<0>(seeds),
              idx,
              std::get<1>(seeds),
              &state);

  int grid_stride = blockDim.x * gridDim.x * unroll_factor;
  int rounded_size = ((numel - 1) / grid_stride + 1) * grid_stride;
  double range = upper - lower;

  for (int linear_index = idx; linear_index < rounded_size; linear_index += grid_stride) {
    auto rand = random_func(&state);

    // ensure that (&rand.x)[ii] is safe
    static_assert(sizeof(rand)/sizeof(rand.x) == unroll_factor, "");

    #pragma unroll
    for (int ii = 0; ii < unroll_factor; ii++) {
      int li = linear_index + blockDim.x * gridDim.x * ii;
      if (li >= numel) {
        continue;
      }
      scalar_t r = static_cast<scalar_t>((&rand.x)[ii]);
      r = r * range + lower;
      if (input[li] <= 0) {
        output[li] = input[li] * r;
        noise[li] = r;
      } else {
        output[li] = input[li];
        noise[li] = static_cast<scalar_t>(0);
      }
    }
    __syncthreads();
  }
}

template <typename scalar_t>
inline void _rrelu_with_noise_cuda_train(
    Tensor& output,
    const Tensor& input_,
    const Tensor& noise_,
    const Scalar& lower_,
    const Scalar& upper_,
    c10::optional<Generator> generator) {
  auto input = input_.contiguous();
  auto noise = noise_.contiguous();
  Tensor tmp_output = output.contiguous();

  int64_t numel = input.numel();
  auto execution_policy = calc_execution_policy(numel);

  auto counter_offset = std::get<0>(execution_policy);
  auto grid = std::get<1>(execution_policy);
  auto block = std::get<2>(execution_policy);

  auto gen = get_generator_or_default<CUDAGeneratorImpl>(
      generator, cuda::detail::getDefaultCUDAGenerator());
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(counter_offset);
  }

  scalar_t* input_data = input.data_ptr<scalar_t>();
  scalar_t* noise_data = noise.data_ptr<scalar_t>();
  scalar_t* output_data = tmp_output.data_ptr<scalar_t>();

  double lower = lower_.to<double>();
  double upper = upper_.to<double>();

  auto stream = at::cuda::getCurrentCUDAStream();

  if (std::is_same<scalar_t, double>::value) {
    rrelu_with_noise_cuda_kernel<scalar_t, 2><<<grid, block, 0, stream>>>(
        numel,
        rng_engine_inputs,
        output_data,
        input_data,
        noise_data,
        lower,
        upper,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) {
          return hiprand_uniform2_double(state);
        });
        C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    // half and float
    rrelu_with_noise_cuda_kernel<scalar_t, 4><<<grid, block, 0, stream>>>(
        numel,
        rng_engine_inputs,
        output_data,
        input_data,
        noise_data,
        lower, upper,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) {
          return hiprand_uniform4(state);
        });
        C10_CUDA_KERNEL_LAUNCH_CHECK();
  }

  if (!output.is_contiguous()) {
    output.copy_(tmp_output);
  }
}

Tensor& rrelu_with_noise_out_cuda(const Tensor& self,
    const Tensor& noise,
    const Scalar& lower,
    const Scalar& upper,
    bool training,
    c10::optional<Generator> generator,
    Tensor& output) {
  TensorArg self_arg{self, "self", 1}, noise_arg{noise, "noise", 2},
      output_arg{output, "output", 3};
  checkAllSameGPU("rrelu_with_noise_out_cuda", {self_arg, noise_arg, output_arg});

  if (training) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        self.scalar_type(), "rrelu_with_noise_out_cuda", [&] {
          _rrelu_with_noise_cuda_train<scalar_t>(
              output, self, noise, lower, upper, generator);
        });
  }
  else {
    auto lower_tensor = lower.to<double>();
    auto upper_tensor = upper.to<double>();
    Scalar negative_slope = (lower_tensor + upper_tensor) / 2;
    at::leaky_relu_out(output, self, negative_slope);
  }
  return output;
}

Tensor rrelu_with_noise_cuda(
    const Tensor& self,
    const Tensor& noise,
    const Scalar& lower,
    const Scalar& upper,
    bool training,
    c10::optional<Generator> generator) {
  Tensor output = at::empty_like(self, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  return at::native::rrelu_with_noise_out_cuda(self, noise, lower, upper, training, generator, output);
}

Tensor& rrelu_with_noise_cuda_(
    Tensor& self,
    const Tensor& noise,
    const Scalar& lower,
    const Scalar& upper,
    bool training,
    c10::optional<Generator> generator) {
  return at::native::rrelu_with_noise_out_cuda(
      self, noise, lower, upper, training, generator, self);
}

// -----------------------------------
// hardshrink
// -----------------------------------
void hardshrink_kernel(TensorIteratorBase& iter, const Scalar& value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardshrink_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return (a >= -lambd && a <= lambd) ? scalar_t(0) : a;
    });
  });
}

void softshrink_kernel(TensorIteratorBase& iter, const Scalar& value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softshrink_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > lambd ? a - lambd : (a < -lambd ? a + lambd : scalar_t(0));
    });
  });
}

void shrink_backward_kernel(TensorIteratorBase& iter, const Scalar& value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "shrink_backward_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t grad_val, scalar_t self_val) -> scalar_t {
      return (self_val >= -lambd && self_val <= lambd) ? scalar_t(0) : grad_val;
    });
  });
}

void hardtanh_backward_kernel(TensorIterator& iter, const Scalar& min, const Scalar& max) {
  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, iter.dtype(), "hardtanh_backward_cuda", [&]() {
    auto min_val = min.to<scalar_t>();
    auto max_val = max.to<scalar_t>();
    gpu_kernel(iter, [min_val, max_val]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return (b <= min_val) || (b >= max_val) ? scalar_t(0) : a;
    });
  });
}

void softplus_kernel(TensorIteratorBase& iter, const Scalar& beta_, const Scalar& threshold_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softplus_cuda", [&]() {
    auto beta = beta_.to<scalar_t>();
    auto threshold = threshold_.to<scalar_t>();
    gpu_kernel(iter, [beta, threshold]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return (a * beta) > threshold ? a : static_cast<scalar_t>(::log1p(std::exp(a * beta))) / beta;
    });
  });
}

void softplus_backward_kernel(TensorIteratorBase& iter, const Scalar& beta_, const Scalar& threshold_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softplus_backward_cuda", [&]() {
    auto beta = beta_.to<scalar_t>();
    auto threshold = threshold_.to<scalar_t>();
    gpu_kernel(iter, [beta, threshold]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      scalar_t z = std::exp(b * beta);
      return (b * beta) > threshold ? a : a * z / (z + scalar_t(1.));
    });
  });
}

template <typename scalar_t>
void threshold_kernel_impl(TensorIteratorBase& iter, scalar_t threshold, scalar_t value) {
  gpu_kernel_with_scalars(iter, [=]GPU_LAMBDA(scalar_t x, scalar_t other) -> scalar_t {
    return x <= threshold ? value : other;
  });
}

static void threshold_kernel_cuda(TensorIteratorBase& iter, const Scalar& threshold, const Scalar& value) {
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "threshold_cuda", [&] {
    threshold_kernel_impl<scalar_t>(iter, threshold.to<scalar_t>(), value.to<scalar_t>());
  });
}

void elu_kernel(TensorIteratorBase& iter, const Scalar& alpha, const Scalar& scale, const Scalar& input_scale) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "elu_cuda", [&]() {
    auto negcoef = alpha.to<scalar_t>() * scale.to<scalar_t>();
    auto poscoef = scale.to<scalar_t>();
    auto negiptcoef = input_scale.to<scalar_t>();
    gpu_kernel(iter, [negcoef, poscoef, negiptcoef]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > scalar_t(0) ? a * poscoef : (static_cast<scalar_t>(std::exp(a * negiptcoef)) - scalar_t(1.)) * negcoef;
    });
  });
}

void elu_backward_kernel(TensorIteratorBase& iter, const Scalar& alpha, const Scalar& scale, const Scalar& input_scale, bool is_result) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "elu_backward_cuda", [&]() {
    auto negcoef = alpha.to<scalar_t>() * scale.to<scalar_t>();
    auto poscoef = scale.to<scalar_t>();
    auto negiptcoef = input_scale.to<scalar_t>();
    gpu_kernel(iter, [negcoef, poscoef, negiptcoef, is_result]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      if (is_result) {
        return b <= scalar_t(0) ? a * negiptcoef * (b + negcoef) : a * poscoef;
      } else {
        return b <= scalar_t(0) ? a * negiptcoef * negcoef * (static_cast<scalar_t>(std::exp(b * negiptcoef))) : a * poscoef;
      }
    });
  });
}

namespace {

void GeluCUDAKernelImpl(TensorIteratorBase& it, bool approximate) {
  if (approximate) {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, it.dtype(), "GeluCUDAKernelImpl", [&]() {
      using T_ACC = acc_type<scalar_t, true>;
      gpu_kernel(it, [] GPU_LAMBDA(scalar_t x) -> scalar_t {
        constexpr T_ACC kBeta = M_SQRT2 * M_2_SQRTPI * T_ACC(0.5);
        constexpr T_ACC kKappa = 0.044715;
        auto inner = kBeta * (static_cast<T_ACC>(x) + kKappa * c10::cuda::compat::pow(static_cast<T_ACC>(x), T_ACC(3)));
        return T_ACC(0.5) * static_cast<T_ACC>(x) * (T_ACC(1) + c10::cuda::compat::tanh(inner));
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, it.dtype(), "GeluCUDAKernelImpl", [&]() {
      using T_ACC = acc_type<scalar_t, true>;
      gpu_kernel(it, [] GPU_LAMBDA(scalar_t x) -> scalar_t {
        return static_cast<T_ACC>(x) *
            c10::cuda::compat::normcdf(static_cast<T_ACC>(x));
      });
    });
  }
}

void GeluBackwardCUDAKernelImpl(TensorIteratorBase& it, bool approximate) {
  if (approximate) {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
        it.dtype(), "GeluBackwardCUDAKernelImpl", [&]() {
          using T_ACC = acc_type<scalar_t, true>;
          gpu_kernel(it, [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
            constexpr T_ACC kBeta = M_SQRT2 * M_2_SQRTPI * T_ACC(0.5);
            constexpr T_ACC kKappa = 0.044715;
            auto inner = kBeta * (static_cast<T_ACC>(x) + kKappa * c10::cuda::compat::pow(static_cast<T_ACC>(x), T_ACC(3)));
            auto tanh_inner = c10::cuda::compat::tanh(inner);

            auto left = T_ACC(0.5) * static_cast<T_ACC>(x);
            auto right = T_ACC(1) + tanh_inner;

            auto left_derivative = 0.5 * right;

            auto tanh_derivative = T_ACC(1) - tanh_inner * tanh_inner;
            auto x_sq = static_cast<T_ACC>(x) * static_cast<T_ACC>(x);
            auto inner_derivative = kBeta * (T_ACC(1) + T_ACC(3) * kKappa * x_sq);
            auto right_derivative = left * tanh_derivative * inner_derivative;

            return static_cast<T_ACC>(dy) * (left_derivative + right_derivative);
        });
      });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
        it.dtype(), "GeluBackwardCUDAKernelImpl", [&]() {
          using T_ACC = acc_type<scalar_t, true>;
          gpu_kernel(it, [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
            constexpr T_ACC kBeta = M_2_SQRTPI * M_SQRT1_2 * T_ACC(0.5);
            const T_ACC cdf = c10::cuda::compat::normcdf(static_cast<T_ACC>(x));
            const T_ACC pdf =
                c10::cuda::compat::exp(
                    T_ACC(-0.5) * static_cast<T_ACC>(x) * static_cast<T_ACC>(x)) *
                kBeta;
            return static_cast<T_ACC>(dy) * (cdf + static_cast<T_ACC>(x) * pdf);
          });
        });
  }
}

void leaky_relu_kernel(TensorIteratorBase& iter, const Scalar& negval_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "leaky_relu_cuda", [&]() {
    auto negval = negval_.to<scalar_t>();
    gpu_kernel(iter, [negval]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > scalar_t(0) ? a : a * negval;
    });
  });
}

void leaky_relu_backward_kernel(TensorIteratorBase& iter, const Scalar& negval_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "leaky_relu_backward_cuda", [&]() {
    auto negval = negval_.to<scalar_t>();
    gpu_kernel(iter, [negval]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a > scalar_t(0) ? b : b * negval;
    });
  });
}

void hardswish_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardswish_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    const T_ACC three(3.0f);
    const T_ACC six(6.0f);
    gpu_kernel(iter, [zero, one_sixth, three, six]GPU_LAMBDA(scalar_t self_val) -> scalar_t {
      T_ACC x = static_cast<T_ACC>(self_val);
      return x * std::min(std::max(x + three, zero), six) * one_sixth;
    });
  });
}

void hardswish_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardswish_backward_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC three(3.0f);
    const T_ACC neg_three(-3.0f);
    const T_ACC one_half(0.5f);
    gpu_kernel(
      iter,
      [zero, three, neg_three, one_half]GPU_LAMBDA(scalar_t grad_val_, scalar_t self_val_) -> scalar_t {
        T_ACC grad_val = static_cast<T_ACC>(grad_val_);
        T_ACC self_val = static_cast<T_ACC>(self_val_);
        if (self_val < neg_three) {
          return zero;
        } else if (self_val <= three) {
          return grad_val * ((self_val / three) + one_half);
        } else {
          return grad_val;
        }
    });
  });
}

void hardsigmoid_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardsigmoid_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    const T_ACC three(3.0f);
    const T_ACC six(6.0f);
    gpu_kernel(iter, [zero, one_sixth, three, six]GPU_LAMBDA(scalar_t self_val) -> scalar_t {
      T_ACC x = static_cast<T_ACC>(self_val);
      return std::min(std::max(x + three, zero), six) * one_sixth;
    });
  });
}

void hardsigmoid_backward_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardsigmoid_backward_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC three(3.0f);
    const T_ACC neg_three(-3.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    gpu_kernel(
      iter,
      [zero, three, neg_three, one_sixth]GPU_LAMBDA(scalar_t grad_val_, scalar_t self_val_) -> scalar_t {
        T_ACC grad_val = static_cast<T_ACC>(grad_val_);
        T_ACC self_val = static_cast<T_ACC>(self_val_);
        return (self_val > neg_three && self_val < three)
          ? grad_val * one_sixth
          : zero;
    });
  });
}

void silu_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "silu_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t x) -> scalar_t {
              using T_ACC = acc_type<scalar_t, true>;
              const T_ACC x_acc = static_cast<T_ACC>(x);
              return x_acc / (T_ACC(1) + c10::cuda::compat::exp(-x_acc));
            });
      });
}

void silu_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "silu_backward_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
              using T_ACC = acc_type<scalar_t, true>;
              const T_ACC dy_acc = static_cast<T_ACC>(dy);
              const T_ACC x_acc = static_cast<T_ACC>(x);
              const T_ACC s_acc =
                  T_ACC(1) / (T_ACC(1) + c10::cuda::compat::exp(-x_acc));
              return dy_acc * s_acc * (T_ACC(1) + x_acc * (T_ACC(1) - s_acc));
            });
      });
}

void mish_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "mish_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t x) -> scalar_t {
          using T_ACC = acc_type<scalar_t, true>;
          const T_ACC x_acc = static_cast<T_ACC>(x);
          return x_acc * c10::cuda::compat::tanh(c10::cuda::compat::log1p(c10::cuda::compat::exp(x_acc)));
      });
      });
}

void mish_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "mish_backward_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
          using T_ACC = acc_type<scalar_t, true>;
          const T_ACC dy_acc = static_cast<T_ACC>(dy);
          const T_ACC x_acc = static_cast<T_ACC>(x);
          const T_ACC s_acc =
              T_ACC(1) / (T_ACC(1) + c10::cuda::compat::exp(-x_acc));
          const T_ACC t_acc =
              c10::cuda::compat::tanh(c10::cuda::compat::log1p(c10::cuda::compat::exp(x_acc)));
          return dy_acc * (t_acc + x_acc * s_acc * (T_ACC(1) - t_acc * t_acc));
      });
      });
}

} // namespace

TORCH_IMPL_FUNC(gelu_out_cuda) (
  const Tensor& self, bool approximate, const Tensor& result
) {
  GeluCUDAKernelImpl(*this, approximate);
}

TORCH_IMPL_FUNC(gelu_backward_out_cuda) (
  const Tensor& grad_output, const Tensor& self, bool approximate, const Tensor& grad_input
) {
  GeluBackwardCUDAKernelImpl(*this, approximate);
}

REGISTER_DISPATCH(hardtanh_backward_stub, &hardtanh_backward_kernel);
REGISTER_DISPATCH(hardshrink_stub, &hardshrink_kernel);
REGISTER_DISPATCH(softshrink_stub, &softshrink_kernel);
REGISTER_DISPATCH(shrink_backward_stub, &shrink_backward_kernel);
REGISTER_DISPATCH(elu_stub, &elu_kernel);
REGISTER_DISPATCH(elu_backward_stub, &elu_backward_kernel);
REGISTER_DISPATCH(leaky_relu_stub, &leaky_relu_kernel);
REGISTER_DISPATCH(leaky_relu_backward_stub, &leaky_relu_backward_kernel);
REGISTER_DISPATCH(hardswish_stub, &hardswish_kernel);
REGISTER_DISPATCH(hardswish_backward_stub, &hardswish_backward_kernel);
REGISTER_DISPATCH(hardsigmoid_stub, &hardsigmoid_kernel);
REGISTER_DISPATCH(hardsigmoid_backward_stub, &hardsigmoid_backward_kernel);
REGISTER_DISPATCH(softplus_stub, &softplus_kernel);
REGISTER_DISPATCH(softplus_backward_stub, &softplus_backward_kernel);
REGISTER_DISPATCH(silu_stub, &silu_kernel);
REGISTER_DISPATCH(silu_backward_stub, &silu_backward_kernel);
REGISTER_DISPATCH(mish_stub, &mish_kernel);
REGISTER_DISPATCH(mish_backward_stub, &mish_backward_kernel);
REGISTER_DISPATCH(threshold_stub, &threshold_kernel_cuda);

} // namespace native
} // namespace at
